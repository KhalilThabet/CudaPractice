
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void doubleElements(int* List)
{
  int i;
  i = threadIdx.x;
  List[i]=1+i;
}


int main()
{
  int N = 5;
  int *List;

  size_t size = N * sizeof(int);

  hipMallocManaged(&List, size);


  size_t threads_per_block = 5;

  doubleElements<<<1, threads_per_block>>>(List);
  hipDeviceSynchronize();
  for (int i=0;i<N;i++) printf("%d \t",List[i]);

  hipFree(List);
}