
#include <hip/hip_runtime.h>
#include <stdio.h>
void init(int *List, int n)
{
    for (int i = 0; i < n; i++)
        List[i] = 0;
}
__global__ void Fibonacci(int *List)
{
    int i;
    i = threadIdx.x;
    if (i > 1)
    {
        List[i] = List[i - 2] + List[i - 1];
    }
}

int main()
{
    int N = 10;
    int *List;

    size_t size = N * sizeof(int);

    hipMallocManaged(&List, size);
    init(List, N);
    List[0] = 1;
    List[1] = 2;
    size_t threads_per_block = 10;

    Fibonacci<<<1, threads_per_block>>>(List);
    hipDeviceSynchronize();
    for (int i = 0; i < N; i++)
        printf("%d \t", List[i]);

    hipFree(List);
}