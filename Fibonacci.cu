#include "hip/hip_runtime.h"
#include <stdio.h>
void init(int *List, int n)
{
    for (int i = 0; i < n; i++)
        List[i] = 0;
}
__global__ void Fibonacci(int *List,int* mutex)
{
    int i;
    i = threadIdx.x;
    if (i > 1) //Parrallel Programming getting results for parrallel summation
    {
        mutex(&lock);
        List[i] = List[i - 2] + List[i - 1];
        mutex(&unlock);
    }
}

int main()
{
    int N = 10;
    int *List;
    int *mutex; //all threads share on mutex.
    hipMallocManaged((void **)&mutex, sizeof(int));
    *mutex = 0;

    size_t size = N * sizeof(int);

    hipMallocManaged(&List, size);
    init(List, N);
    List[0] = 1;
    List[1] = 2;
    size_t threads_per_block = 10;

    Fibonacci<<<1, threads_per_block>>>(List);
    hipDeviceSynchronize();
    for (int i = 0; i < N; i++)
        printf("%d \t", List[i]);

    hipFree(List);
    hipFree(mutex);
}