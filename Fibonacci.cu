#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__device__ void lock(int *mutex)
{
    while (atomicCAS(mutex, 0, 1) != 0)
        ;
}
__device__ void unlock(int *mutex)
{
    atomicExch(mutex, 0);
}

void init(int *List, int n)
{
    for (int i = 0; i < n; i++)
        List[i] = 0;
}
__global__ void Fibonacci(int *List,int* mutex)
{
    int i;
    i = threadIdx.x;
    if (i > 1) //Parrallel Programming getting results for parrallel summation
    {
        lock(mutex);
        List[i] = List[i - 2] + List[i - 1];
        unlock(mutex);
        
    }
}

int main()
{
    int N = 10;
    int *List;
    int *mutex; //all threads share on mutex.
    hipMallocManaged((void **)&mutex, sizeof(int));
    *mutex = 0;

    size_t size = N * sizeof(int);

    hipMallocManaged(&List, size);
    init(List, N);
    List[0] = 1;
    List[1] = 2;
    size_t threads_per_block = 10;

    Fibonacci<<<1, threads_per_block>>>(List);
    hipDeviceSynchronize();
    for (int i = 0; i < N; i++)
        printf("%d \t", List[i]);

    hipFree(List);
    hipFree(mutex);
}