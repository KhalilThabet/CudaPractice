
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void Fibonacci(int *List)
{
    int i;
    i = threadIdx.x;
    if (i > 1)
        List[i] = List[i - 1] + List[i - 2];
}

int main()
{
    int N = 10;
    int *List;

    size_t size = N * sizeof(int);

    hipMallocManaged(&List, size);
    List[0] = 1;
    List[1] = 1;
    size_t threads_per_block = 10;

    Fibonacci<<<1, threads_per_block>>>(List);
    hipDeviceSynchronize();
    for (int i = 0; i < N; i++)
        printf("%d \t", List[i]);

    hipFree(List);
}