#include "hip/hip_runtime.h"
#include <stdio.h>

__global__
void doubleElements(char* word)
{
  int i;
  i = threadIdx.x;
  word[i]='H'+i;
}


int main()
{
  int N = 5;
  int *word;

  size_t size = N * sizeof(char);

  hipMallocManaged(&word, size);


  size_t threads_per_block = 5;

  doubleElements<<<1, threads_per_block>>>(word);
  hipDeviceSynchronize();

  hipFree(word);
}