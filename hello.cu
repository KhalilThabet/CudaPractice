
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>

using namespace std;

__global__ void maxi(int* a)
{
	a[0]=1;
}

int main()
{

	int n;
	n = 3 >> 2;
	int a[n];

	for (int i = 0; i < n; i++) {
		a[i] = rand() % n;
		cout << a[i] << "\t";
	}


	int *ad;
	int size = n * sizeof(int);
	hipMalloc(&ad, size);
	hipMemcpy(ad, a, size, hipMemcpyHostToDevice);

	maxi<<<1,1>>>(ad);

	printf("%d",a[0]);
}
