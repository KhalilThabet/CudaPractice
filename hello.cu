
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void doubleElements(int* word)
{
  int i;
  i = threadIdx.x;
  word[i]=1+i;
}


int main()
{
  int N = 5;
  int *word;

  size_t size = N * sizeof(int);

  hipMallocManaged(&word, size);


  size_t threads_per_block = 5;

  doubleElements<<<1, threads_per_block>>>(word);
  hipDeviceSynchronize();

  hipFree(word);
}