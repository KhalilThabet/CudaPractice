
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void doubleElements(int* word)
{
  int i;
  i = threadIdx.x;
  word[i]=1+i;
}


int main()
{
  int N = 5;
  int *word;

  size_t size = N * sizeof(int);

  hipMallocManaged(&word, size);


  size_t threads_per_block = 5;

  doubleElements<<<1, threads_per_block>>>(word);
  hipDeviceSynchronize();
  for (int i=0;i<N;i++) printf("%d \t",word[i]);

  hipFree(word);
}